#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "error-check.hpp"
#include "matrix.cuh"

#define ITER_CHECK 25     // status printed and convergence check every ITER_CHECK iterations
#define MAX_ITER 200      // max number of iterations
#define CONVERGE_THRESH 0 // set to zero to guarantee MAX_ITER iterations, 0.001 is a good value otherwise

void update_div(
    Matrix W0, Matrix H0, Matrix X0, const uint32_t M, const uint32_t K, const uint32_t N, const float thresh,
    const int32_t max_iter, int32_t verbose, hipStream_t stream
);
uint32_t nextpow2(uint32_t x);
Matrix read_matrix(std::string file, hipStream_t stream);
void write_matrix(Matrix A_padded, uint32_t rows, uint32_t cols, std::string file);


int32_t main(int32_t argc, char *argv[]) {
    hipStream_t stream = NULL;

    Matrix W0 = read_matrix("../W.bin", stream);
    Matrix X0 = read_matrix("../X.bin", stream);
    Matrix H0 = read_matrix("../H.bin", stream);

    // make sure no zero elements
    matrix_eps_d(X0, 128, stream);
    matrix_eps_d(H0, 128, stream);
    matrix_eps_d(W0, 128, stream);

    const uint32_t M = W0.rows;
    const uint32_t K = W0.cols;
    const uint32_t N = H0.cols;

    // matrices to hold padded versions of matrices
    Matrix W(0.0f, M, K, true);
    Matrix H(0.0f, K, N, true);
    Matrix X(0.0f, M, N, true);

    W0.copy_to_padded(&W);
    H0.copy_to_padded(&H);
    X0.copy_to_padded(&X);

    // iterative nmf minimization
    update_div(W, H, X, M, K, N, CONVERGE_THRESH, MAX_ITER, 1, stream);

    // write results matrices to binary files
    // (can be read with export_bin.m in Matlab)
    write_matrix(W, W0.rows, W0.cols, "../Wout.bin");
    write_matrix(H, H0.rows, H0.cols, "../Hout.bin");

    return 0;
}

void init_params(uint32_t value, uint32_t *params) {
    uint32_t padded_value = value;
    if(value % PAD_MULT != 0) {
        padded_value = value + (PAD_MULT - (value % PAD_MULT));
    }

    uint32_t rem;
    rem = nextpow2(padded_value / 128 + (!(padded_value % 128) ? 0 : 1));
    if(rem <= 128) {
        params[0] = 128;
        params[1] = rem;
    } else if(rem <= 512) {
        params[0] = rem;
        params[1] = 128;
    } else {
        fprintf(stderr, "reduction parameter error\n");
        exit(1);
    }

    params[2] = 1;
    params[3] = 1;
}

void update_div(
    Matrix W, Matrix H, Matrix X, const uint32_t M, const uint32_t K, const uint32_t N, const float thresh,
    const int32_t max_iter, int32_t verbose, hipStream_t stream
) {
    // run iterative multiplicative updates on W,H

    cublasInit();

    // find reduction parameters
    uint32_t N_params[4]; // N size reductions (rows)
    uint32_t M_params[4]; // M size reductions (cols)

    init_params(N, N_params);
    init_params(M, M_params);

    // block size in vector arithmetic operations
    const int32_t BLOCK_SIZE = 128;

    // initialize temp matrices -----------------------
    Matrix Z(0.0f, M, N, true);     // Matrix to hold X./(W*H+EPS)
    Matrix WtZ(0.0f, K, N, true);   // Matrix to hold W'*Z
    Matrix ZHt(0.0f, M, K, true);   // Matrix to hold Z*H'
    Matrix sumW(0.0f, 1, K, true);  // Matrix to hold sum(W) [sum of cols of W]
    Matrix sumH2(0.0f, K, 1, true); // Matrix to hold sum(H,2) [sum of rows of H]

    for(int32_t i = 0; i < max_iter; i++) {
        /* matlab algorithm
           Z = X./(W*H+eps); H = H.*(W'*Z)./(repmat(sum(W)',1,F));
           Z = X./(W*H+eps);
           W = W.*(Z*H')./(repmat(sum(H,2)',N,1));
           */

        //
        // UPDATE H -----------------------------
        //

        // WH = W*H
        matrix_multiply_d(W, H, Z);

        // WH = WH+EPS
        matrix_eps_d(Z, BLOCK_SIZE, stream);

        // Z = X./WH
        element_divide_d(X, Z, Z, BLOCK_SIZE);

        // sum cols of W into row vector
        sum_cols_d(compute, W, sumW, M_params);
        matrix_eps_d(sumW, 32, stream);

        // convert sumW to col vector (transpose)
        sumW.rows = sumW.cols;
        sumW.cols = 1;

        // WtZ = W'*Z
        matrix_multiply_AtB_d(W, Z, WtZ);

        // WtZ = WtZ./(repmat(sum(W)',1,H.cols)
        //[element divide cols of WtZ by sumW']
        col_divide_d(WtZ, sumW, WtZ);

        // H = H.*WtZ
        element_multiply_d(H, WtZ, H, BLOCK_SIZE);

        //
        // UPDATE W ---------------------------
        //

        // WH = W*H
        matrix_multiply_d(W, H, Z);

        // WH = WH+EPS
        matrix_eps_d(Z, BLOCK_SIZE, stream);

        // Z = X./WH
        element_divide_d(X, Z, Z, BLOCK_SIZE);

        // sum rows of H into col vector
        sum_rows_d(compute, H, sumH2, N_params);
        matrix_eps_d(sumH2, 32, stream);

        // convert sumH2 to row vector (transpose)
        sumH2.cols = sumH2.rows;
        sumH2.rows = 1;

        // ZHt = Z*H'
        matrix_multiply_ABt_d(Z, H, ZHt);

        // ZHt = ZHt./(repmat(sum(H,2)',W.rows,1)
        //[element divide rows of ZHt by sumH2']
        row_divide_d(ZHt, sumH2, ZHt);

        // W = W.*ZHt
        element_multiply_d(W, ZHt, W, BLOCK_SIZE);

        // reset sumW to row vector
        sumW.cols = sumW.rows;
        sumW.rows = 1;
        // reset sumH2 to col vector
        sumH2.rows = sumH2.cols;
        sumH2.cols = 1;
    }

    // clean up extra reduction memory
    sum_cols_d(cleanup, W, sumW, M_params);
    sum_rows_d(cleanup, H, sumH2, N_params);

    cublasShutdown();
}

uint32_t nextpow2(uint32_t x) {
    x = x - 1;
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    return x + 1;
}

Matrix read_matrix(std::string file, hipStream_t stream) {
    // read Matrix in from file, store in column-major order

    FILE *fp;
    size_t count;

    uint32_t rows, cols;

    fp = fopen(file.c_str(), "rb");
    count = fread(&rows, sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "read_matrix: fread error\n");
    count = fread(&cols, sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "read_matrix: fread error\n");

    size_t size = rows * cols;
    float *temp = (float *) malloc(sizeof(float) * size);
    count = fread(temp, sizeof(float), size, fp);
    if(count < size) fprintf(stderr, "read_matrix: fread error\n");
    fclose(fp);

    Matrix A(temp, rows, cols, false);

    free(temp);

    printf("read %s [%ix%i]\n", file.c_str(), A.rows, A.cols);

    return A;
}

void write_matrix(Matrix A_padded, uint32_t rows, uint32_t cols, std::string file) {
    // write Matrix to file using column-major order
    // dimensions are written as leading ints

    assert(rows <= A_padded.rows);
    assert(cols <= A_padded.cols);

    float *temp;
    cudaAssert(hipHostMalloc((void **) &temp, rows * cols * sizeof(float)));
    hipMemcpy2D(
        temp, sizeof(float) * rows, A_padded.data, sizeof(float) * A_padded.rows, sizeof(float) * rows, cols,
        hipMemcpyDeviceToHost
    );

    FILE *fp;
    size_t count;

    fp = fopen(file.c_str(), "wb");
    count = fwrite(&(rows), sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "write_matrix: fwrite error\n");
    count = fwrite(&(cols), sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "write_matrix: fwrite error\n");

    count = fwrite(temp, sizeof(float), rows * cols, fp);
    if(count < (size_t) (rows * cols)) fprintf(stderr, "write_matrix: fwrite error\n");
    fclose(fp);

    cudaAssert(hipHostFree(temp));

    printf("write %s [%ix%i]\n", file.c_str(), rows, cols);
}
