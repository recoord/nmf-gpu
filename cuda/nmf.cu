#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "error-check.hpp"
#include "matrix.cuh"

#define ITER_CHECK 25     // status printed and convergence check every ITER_CHECK iterations
#define MAX_ITER 200      // max number of iterations
#define CONVERGE_THRESH 0 // set to zero to guarantee MAX_ITER iterations, 0.001 is a good value otherwise

void update_div(
    Matrix W0, Matrix H0, Matrix X0, const float thresh, const int32_t max_iter, int32_t verbose, hipStream_t stream
);
uint32_t nextpow2(uint32_t x);
Matrix read_matrix(std::string file, hipStream_t stream);
void write_matrix(Matrix A, std::string file);


int32_t main(int32_t argc, char *argv[]) {
    hipStream_t stream = NULL;

    Matrix W = read_matrix("../W.bin", stream);
    Matrix X = read_matrix("../X.bin", stream);
    Matrix H = read_matrix("../H.bin", stream);

    // make sure no zero elements
    matrix_eps_d(X, 128, stream);
    matrix_eps_d(H, 128, stream);
    matrix_eps_d(W, 128, stream);

    // iterative nmf minimization
    update_div(W, H, X, CONVERGE_THRESH, MAX_ITER, 1, stream);

    // write results matrices to binary files
    // (can be read with export_bin.m in Matlab)
    write_matrix(W, "../Wout.bin");
    write_matrix(H, "../Hout.bin");

    return 0;
}


void update_div(
    Matrix W0, Matrix H0, Matrix X0, const float thresh, const int32_t max_iter, int32_t verbose, hipStream_t stream
) {
    // run iterative multiplicative updates on W,H

    cublasInit();

    const uint32_t M = W0.rows;
    const uint32_t K = W0.cols;
    const uint32_t N = H0.cols;

    // pad Matrix dimensions to multiples of:
    const uint32_t PAD_MULT = 32;

    uint32_t M_padded = M;
    if(M % PAD_MULT != 0) M_padded = M + (PAD_MULT - (M % PAD_MULT));

    uint32_t K_padded = K;
    if(K % PAD_MULT != 0) K_padded = K + (PAD_MULT - (K % PAD_MULT));

    uint32_t N_padded = N;
    if(N % PAD_MULT != 0) N_padded = N + (PAD_MULT - (N % PAD_MULT));

    // find reduction parameters
    uint32_t N_params[4] = {1, 1, 1, 1}; // N size reductions (rows)
    uint32_t M_params[4] = {1, 1, 1, 1}; // M size reductions (cols)

    uint32_t rem;
    rem = nextpow2(N_padded / 128 + (!(N_padded % 128) ? 0 : 1));
    if(rem <= 128) {
        N_params[0] = 128;
        N_params[1] = rem;
    } else if(rem <= 512) {
        N_params[0] = rem;
        N_params[1] = 128;
    } else {
        fprintf(stderr, "reduction parameter error\n");
        exit(1);
    }

    rem = nextpow2(M_padded / 128 + (!(M_padded % 128) ? 0 : 1));
    if(rem <= 128) {
        M_params[0] = 128;
        M_params[1] = rem;
    } else if(rem <= 512) {
        M_params[0] = rem;
        M_params[1] = 128;
    } else {
        fprintf(stderr, "reduction parameter error\n");
        exit(1);
    }

    // block size in vector arithmetic operations
    const int32_t BLOCK_SIZE = 128;

    // initialize temp matrices -----------------------

    // TODO: Why can't the other matrices be initialized with padding=true?

    Matrix Z(0.0f, M, N, true);                  // Matrix to hold X./(W*H+EPS)
    Matrix WtZ(0.0f, K_padded, N_padded, false); // Matrix to hold W'*Z
    Matrix ZHt(0.0f, M_padded, K_padded, false); // Matrix to hold Z*H'
    Matrix sumW(0.0f, 1, K_padded, false);       // Matrix to hold sum(W) [sum of cols of W]
    Matrix sumH2(0.0f, K_padded, 1, false);      // Matrix to hold sum(H,2) [sum of rows of H]

    // matrices to hold padded versions of matrices
    Matrix W(0.0f, M, K, true);
    Matrix H(0.0f, K, N, true);
    Matrix X(0.0f, M, N, true);

    W0.copy_to_padded(&W);
    H0.copy_to_padded(&H);
    X0.copy_to_padded(&X);

    for(int32_t i = 0; i < max_iter; i++) {
        /* matlab algorithm
           Z = X./(W*H+eps); H = H.*(W'*Z)./(repmat(sum(W)',1,F));
           Z = X./(W*H+eps);
           W = W.*(Z*H')./(repmat(sum(H,2)',N,1));
           */

        //
        // UPDATE H -----------------------------
        //

        // WH = W*H
        matrix_multiply_d(W, H, Z);

        // WH = WH+EPS
        matrix_eps_d(Z, BLOCK_SIZE, stream);

        // Z = X./WH
        element_divide_d(X, Z, Z, BLOCK_SIZE);

        // sum cols of W into row vector
        sum_cols_d(compute, W, sumW, M_params);
        matrix_eps_d(sumW, 32, stream);

        // convert sumW to col vector (transpose)
        sumW.rows = sumW.cols;
        sumW.cols = 1;

        // WtZ = W'*Z
        matrix_multiply_AtB_d(W, Z, WtZ);

        // WtZ = WtZ./(repmat(sum(W)',1,H.cols)
        //[element divide cols of WtZ by sumW']
        col_divide_d(WtZ, sumW, WtZ);

        // H = H.*WtZ
        element_multiply_d(H, WtZ, H, BLOCK_SIZE);

        //
        // UPDATE W ---------------------------
        //

        // WH = W*H
        matrix_multiply_d(W, H, Z);

        // WH = WH+EPS
        matrix_eps_d(Z, BLOCK_SIZE, stream);

        // Z = X./WH
        element_divide_d(X, Z, Z, BLOCK_SIZE);

        // sum rows of H into col vector
        sum_rows_d(compute, H, sumH2, N_params);
        matrix_eps_d(sumH2, 32, stream);

        // convert sumH2 to row vector (transpose)
        sumH2.cols = sumH2.rows;
        sumH2.rows = 1;

        // ZHt = Z*H'
        matrix_multiply_ABt_d(Z, H, ZHt);

        // ZHt = ZHt./(repmat(sum(H,2)',W.rows,1)
        //[element divide rows of ZHt by sumH2']
        row_divide_d(ZHt, sumH2, ZHt);

        // W = W.*ZHt
        element_multiply_d(W, ZHt, W, BLOCK_SIZE);

        // reset sumW to row vector
        sumW.cols = sumW.rows;
        sumW.rows = 1;
        // reset sumH2 to col vector
        sumH2.rows = sumH2.cols;
        sumH2.cols = 1;
    }

    // copy padded Matrix to unpadded matrices
    W0.copy_from_padded(&W);
    H0.copy_from_padded(&H);

    // clean up extra reduction memory
    sum_cols_d(cleanup, W, sumW, M_params);
    sum_rows_d(cleanup, H, sumH2, N_params);

    cublasShutdown();
}

uint32_t nextpow2(uint32_t x) {
    x = x - 1;
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    return x + 1;
}

Matrix read_matrix(std::string file, hipStream_t stream) {
    // read Matrix in from file, store in column-major order

    FILE *fp;
    size_t count;

    uint32_t rows, cols;

    fp = fopen(file.c_str(), "rb");
    count = fread(&rows, sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "read_matrix: fread error\n");
    count = fread(&cols, sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "read_matrix: fread error\n");

    size_t size = rows * cols;
    float *temp = (float *) malloc(sizeof(float) * size);
    count = fread(temp, sizeof(float), size, fp);
    if(count < size) fprintf(stderr, "read_matrix: fread error\n");
    fclose(fp);

    Matrix A(temp, rows, cols, false);

    free(temp);

    printf("read %s [%ix%i]\n", file.c_str(), A.rows, A.cols);

    return A;
}

void write_matrix(Matrix A, std::string file) {
    // write Matrix to file using column-major order
    // dimensions are written as leading ints

    size_t size = A.rows * A.cols * sizeof(float);
    float *temp;
    cudaAssert(hipHostMalloc((void **) &temp, size));
    cudaAssert(hipMemcpy(temp, A.data, size, hipMemcpyDeviceToHost));

    FILE *fp;
    size_t count;

    fp = fopen(file.c_str(), "wb");
    count = fwrite(&(A.rows), sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "write_matrix: fwrite error\n");
    count = fwrite(&(A.cols), sizeof(uint32_t), 1, fp);
    if(count < 1) fprintf(stderr, "write_matrix: fwrite error\n");

    count = fwrite(temp, sizeof(float), A.rows * A.cols, fp);
    if(count < (size_t) (A.rows * A.cols)) fprintf(stderr, "write_matrix: fwrite error\n");
    fclose(fp);

    cudaAssert(hipHostFree(temp));

    printf("write %s [%ix%i]\n", file.c_str(), A.rows, A.cols);
}
